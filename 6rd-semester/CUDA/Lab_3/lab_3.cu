
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

class COV_AVG {
private:
    double3 avg;
    double * cov_matr;
    double * cov_matr_GPU;

public: __host__ COV_AVG(uchar4 * pixels, int numPixels) {
    initAVG(pixels, numPixels);
    initCOV(pixels, numPixels);

    hipMalloc( & cov_matr_GPU, sizeof(double) * 9);
    hipMemcpy(cov_matr_GPU, cov_matr, sizeof(double) * 9, hipMemcpyHostToDevice);
  }

  void initAVG(uchar4 * pixels, int numPixels){
      avg = make_double3(0., 0., 0.);

      for (int i = 0; i < numPixels; i++) {
          avg.x += pixels[i].x;
          avg.y += pixels[i].y;
          avg.z += pixels[i].z;
      }
      avg.x /= numPixels;
      avg.y /= numPixels;
      avg.z /= numPixels;
  }

  void initCOV(uchar4 * pixels, int numPixels) {
      cov_matr = (double * ) malloc(sizeof(double) * 9);
      double3 tmp;
      for (int i = 0; i < numPixels; i++) {
          auto pix = pixels[i];
          tmp = make_double3(pix.x - avg.x, pix.y - avg.y, pix.z - avg.z);
          cov_matr[0] += tmp.x * tmp.x;
          cov_matr[1] += tmp.x * tmp.y;
          cov_matr[2] += tmp.x * tmp.z;
          cov_matr[3] += tmp.y * tmp.x;
          cov_matr[4] += tmp.y * tmp.y;
          cov_matr[5] += tmp.y * tmp.z;
          cov_matr[6] += tmp.z * tmp.x;
          cov_matr[7] += tmp.z * tmp.y;
          cov_matr[8] += tmp.z * tmp.z;
      }

      for (int i = 0; i < 9; i++)
          cov_matr[i] /= (numPixels - 1);
      inverseCOV();
  }

    void inverseCOV() {
        double * tmp = (double * ) malloc(sizeof(double) * 9);

        tmp[0] =   cov_matr[4] * cov_matr[8] - cov_matr[5] * cov_matr[7];
        tmp[1] = -(cov_matr[1] * cov_matr[8] - cov_matr[2] * cov_matr[7]);
        tmp[2] =   cov_matr[1] * cov_matr[5] - cov_matr[2] * cov_matr[4];
        tmp[3] = -(cov_matr[3] * cov_matr[8] - cov_matr[5] * cov_matr[6]);
        tmp[4] =   cov_matr[0] * cov_matr[8] - cov_matr[2] * cov_matr[6];
        tmp[5] = -(cov_matr[0] * cov_matr[5] - cov_matr[2] * cov_matr[3]);
        tmp[6] =   cov_matr[3] * cov_matr[7] - cov_matr[4] * cov_matr[6];
        tmp[7] = -(cov_matr[0] * cov_matr[7] - cov_matr[1] * cov_matr[6]);
        tmp[8] =   cov_matr[0] * cov_matr[4] - cov_matr[1] * cov_matr[3];

        double det = (cov_matr[0] * cov_matr[4] * cov_matr[8]) +
                     (cov_matr[1] * cov_matr[5] * cov_matr[6]) +
                     (cov_matr[2] * cov_matr[3] * cov_matr[7]) -
                     (cov_matr[2] * cov_matr[4] * cov_matr[6]) -
                     (cov_matr[1] * cov_matr[3] * cov_matr[8]) -
                     (cov_matr[0] * cov_matr[5] * cov_matr[7]);

        for (int i = 0; i < 9; i++)
            tmp[i] /= det;
        memcpy(cov_matr, tmp, 9 * sizeof(double));
        free(tmp);
    }


    __device__ double3 getAVG() {
    return avg;
  }

  __device__ double* getCOV() {
    return cov_matr_GPU;
  }

  __host__ ~COV_AVG() {
    free(cov_matr);
  }
};

__global__ void kernel(uchar4 * input_pixels, int len, COV_AVG * cov_avg, int num_classes) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = gridDim.x * blockDim.x;

  for (;idx < len; idx += offset) {
        uchar4 pixel = input_pixels[idx];
        double val = DBL_MAX - 1;
        int class_ind = 0;

        for (int i = 0; i < num_classes; i++) {
            auto avg = cov_avg[i].getAVG();
            auto cov = cov_avg[i].getCOV();
            auto tmp = make_double3(pixel.x - avg.x, pixel.y - avg.y, pixel.z - avg.z);

          double res =  (tmp.x * cov[0] + tmp.y * cov[3] + tmp.z * cov[6]) * tmp.x +
                        (tmp.x * cov[1] + tmp.y * cov[4] + tmp.z * cov[7]) * tmp.y +
                        (tmp.x * cov[2] + tmp.y * cov[5] + tmp.z * cov[8]) * tmp.z;

          if (res < val) {
              val = res;
            class_ind = i;
          }
        }

        input_pixels[idx].w = class_ind;
  }
}

int main() {
    // Считывание данных
    char fileIn[1024];
    char fileOut[1024];
    scanf("%s", fileIn);
    scanf("%s", fileOut);

    int width, height;
    FILE * fp = fopen(fileIn, "rb");
    fread( & width, sizeof(int), 1, fp);
    fread( & height, sizeof(int), 1, fp);

    uchar4* image_bin = (uchar4*) malloc(sizeof(uchar4) * width * height);
    fread(image_bin, sizeof(uchar4), width * height, fp);
    fclose(fp);

    int count_classes;
    scanf("%d", &count_classes);

    COV_AVG*  matrixes = (COV_AVG*) malloc(sizeof(COV_AVG) * count_classes);
    for (int i = 0; i < count_classes; i++) {
        int pixel_count;
        scanf("%d", &pixel_count);

        uchar4* class_data = (uchar4*) malloc(sizeof(uchar4) * pixel_count);
        int x_pos, y_pos;

        for (int j = 0; j < pixel_count; j++) {
            scanf("%d", &x_pos);
            scanf("%d", &y_pos);
            class_data[j] = image_bin[y_pos * width + x_pos];
        }

        matrixes[i] = COV_AVG(class_data, pixel_count);
        free(class_data);
    }

    // Копирование данных на GPU
    uchar4 * image_dev;
    COV_AVG * dev_matrixes;

    hipMalloc( & image_dev, sizeof(uchar4) * width * height);
    hipMemcpy(image_dev, image_bin, sizeof(uchar4) * width * height, hipMemcpyHostToDevice);

    hipMalloc( & dev_matrixes, sizeof(COV_AVG) * count_classes);
    hipMemcpy(dev_matrixes, matrixes, sizeof(COV_AVG) * count_classes, hipMemcpyHostToDevice);

    kernel <<< 32, 32 >>> (image_dev, width * height, dev_matrixes, count_classes);

    hipMemcpy(image_bin, image_dev, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);


    hipFree(image_dev);
    hipFree(dev_matrixes);
    free(matrixes);

    // запись всех данных в файл
    fp = fopen(fileOut, "wb");
    fwrite( & width, sizeof(int), 1, fp);
    fwrite( & height, sizeof(int), 1, fp);
    fwrite(image_bin, sizeof(uchar4), width * height, fp);
    fclose(fp);

    free(image_bin);
    return 0;
}