
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>
#define CSC(call)  									\
do {											\
	hipError_t res = call;							\
	if (res != hipSuccess) {							\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);								\
	}										\
} while(0)


class COV_AVG {
public:
  double3 avg;
  double * cov_matr;
  double * cov_matr_GPU;
  
public:
    COV_AVG(uchar4 * pixels, int numPixels) {
        initAVG(pixels, numPixels);
        initCOV(pixels, numPixels);
        
        CSC(hipMalloc( & cov_matr_GPU, sizeof(double) * 9));
        CSC(hipMemcpy(cov_matr_GPU, cov_matr, sizeof(double) * 9, hipMemcpyHostToDevice));
    }
        
    __device__
    double3 getAVG() {
        return avg;
    }

    __device__
    double* getCOV() {
        return cov_matr_GPU;
    }

    ~COV_AVG() {
        free(cov_matr);
    }
    
    void freeGPU() {
        CSC(hipFree(cov_matr_GPU));
    }
    
private:
    void initAVG(uchar4 * pixels, int numPixels) {
        avg = make_double3(0., 0., 0.);
    
        for (int i = 0; i < numPixels; i++) {
          avg.x += pixels[i].x;
          avg.y += pixels[i].y;
          avg.z += pixels[i].z;
        }
        avg.x /= numPixels;
        avg.y /= numPixels;
        avg.z /= numPixels;
    }
    
    void initCOV(uchar4 * pixels, int numPixels) {
        cov_matr = (double * ) malloc(sizeof(double) * 9);
        double3 tmp;
        for (int i = 0; i < numPixels; i++) {
            auto pix = pixels[i];
            tmp = make_double3(pix.x - avg.x, pix.y - avg.y, pix.z - avg.z);
            cov_matr[0] += tmp.x * tmp.x;
            cov_matr[1] += tmp.x * tmp.y;
            cov_matr[2] += tmp.x * tmp.z;
            cov_matr[3] += tmp.y * tmp.x;
            cov_matr[4] += tmp.y * tmp.y;
            cov_matr[5] += tmp.y * tmp.z;
            cov_matr[6] += tmp.z * tmp.x;
            cov_matr[7] += tmp.z * tmp.y;
            cov_matr[8] += tmp.z * tmp.z;
        }
        
        for (int i = 0; i < 9; i++)
            cov_matr[i] /= (numPixels - 1);
        inverseCOV();
    }
        
    void inverseCOV() {
        double * tmp = (double * ) malloc(sizeof(double) * 9);
        
        tmp[0] =   cov_matr[4] * cov_matr[8] - cov_matr[5] * cov_matr[7];
        tmp[1] = -(cov_matr[1] * cov_matr[8] - cov_matr[2] * cov_matr[7]);
        tmp[2] =   cov_matr[1] * cov_matr[5] - cov_matr[2] * cov_matr[4];
        tmp[3] = -(cov_matr[3] * cov_matr[8] - cov_matr[5] * cov_matr[6]);
        tmp[4] =   cov_matr[0] * cov_matr[8] - cov_matr[2] * cov_matr[6];
        tmp[5] = -(cov_matr[0] * cov_matr[5] - cov_matr[2] * cov_matr[3]);
        tmp[6] =   cov_matr[3] * cov_matr[7] - cov_matr[4] * cov_matr[6];
        tmp[7] = -(cov_matr[0] * cov_matr[7] - cov_matr[1] * cov_matr[6]);
        tmp[8] =   cov_matr[0] * cov_matr[4] - cov_matr[1] * cov_matr[3];
        
        double det = (cov_matr[0] * cov_matr[4] * cov_matr[8]) +
                   (cov_matr[1] * cov_matr[5] * cov_matr[6]) +
                   (cov_matr[2] * cov_matr[3] * cov_matr[7]) -
                   (cov_matr[2] * cov_matr[4] * cov_matr[6]) -
                   (cov_matr[1] * cov_matr[3] * cov_matr[8]) -
                   (cov_matr[0] * cov_matr[5] * cov_matr[7]);
        
        for (int i = 0; i < 9; i++)
          tmp[i] /= det;
        memcpy(cov_matr, tmp, 9 * sizeof(double));
        free(tmp);
    }
};

__global__
void kernel(uchar4 * input_pixels, int len, COV_AVG * cov_avg, int num_classes) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x * blockDim.x;

    for (;idx < len; idx += offset) {
        uchar4 pixel = input_pixels[idx];
        double val = -DBL_MAX;
        int class_ind = 0;
        
        for (int i = 0; i < num_classes; i++) {
            auto avg = cov_avg[i].getAVG();
            auto cov = cov_avg[i].getCOV();
            auto tmp = make_double3(pixel.x - avg.x, pixel.y - avg.y, pixel.z - avg.z);
            
            double res =    (tmp.x * cov[0]  + tmp.y * cov[3] + tmp.z * cov[6]) * -tmp.x +
                            (tmp.x * cov[1]  + tmp.y * cov[4] + tmp.z * cov[7]) * -tmp.y +
                            (tmp.x * cov[2]  + tmp.y * cov[5] + tmp.z * cov[8]) * -tmp.z;
            
            if (res > val) {
                val = res;
                class_ind = i;
            }
        }
//        input_pixels[idx].w = class_ind;
        input_pixels[idx].w = cov_avg[1].getAVG().y;
    }
}

int main() {
    // Считывание данных
    char fileIn[1024];
    char fileOut[1024];
    scanf("%s", fileIn);
    scanf("%s", fileOut);
    
    int width, height;
    FILE * fp = fopen(fileIn, "rb");
    fread( & width, sizeof(int), 1, fp);
    fread( & height, sizeof(int), 1, fp);

    uchar4* image_bin = (uchar4*) malloc(sizeof(uchar4) * width * height);
    fread(image_bin, sizeof(uchar4), width * height, fp);
    fclose(fp);

    int count_classes;
    scanf("%d", &count_classes);


    COV_AVG*  matrixes = (COV_AVG*) malloc(sizeof(COV_AVG) * count_classes);
    for (int i = 0; i < count_classes; i++) {
        int pixel_count;
        scanf("%d", &pixel_count);
        
        uchar4* class_data = (uchar4*) malloc(sizeof(uchar4) * pixel_count);
        int x_pos, y_pos;
        
        for (int j = 0; j < pixel_count; j++) {
            scanf("%d", &x_pos);
            scanf("%d", &y_pos);
            class_data[j] = image_bin[y_pos * width + x_pos];
        }
    
        matrixes[i] = COV_AVG(class_data, pixel_count);
        free(class_data);
    }

    // Копирование данных на GPU
    uchar4 * image_dev;
    COV_AVG * dev_matrixes;

    CSC(hipMalloc( & image_dev, sizeof(uchar4) * width * height));
    CSC(hipMemcpy(image_dev, image_bin, sizeof(uchar4) * width * height, hipMemcpyHostToDevice));

    CSC(hipMalloc( & dev_matrixes, sizeof(COV_AVG) * count_classes));
    CSC(hipMemcpy(dev_matrixes, matrixes, sizeof(COV_AVG) * count_classes, hipMemcpyHostToDevice));
    free(matrixes);

    kernel <<< 1, 1 >>> (image_dev, width * height, dev_matrixes, count_classes);
    CSC(hipGetLastError());
    CSC(hipMemcpy(image_bin, image_dev, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));

    for( int i = 0; i < width * height; i++) {
        fprintf(stderr, "%d ", image_bin[i].w);
    }

    CSC(hipFree(image_dev));
    CSC(hipFree(dev_matrixes));

    // запись всех данных в файл
    FILE * fp2 = fopen(fileOut, "wb");
    fwrite( & width, sizeof(int), 1, fp2);
    fwrite( & height, sizeof(int), 1, fp2);
    fwrite(image_bin, sizeof(uchar4), width * height, fp2);
    fclose(fp2);

    free(image_bin);
    return 0;
}