
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <float.h>

class Matrix {
public:
    int n = 3;
    double* arr;
    double* arr_dev;
    Matrix() {
        arr = (double*)malloc(sizeof(double) * n * n);
        hipMalloc(&arr_dev, sizeof(double)  * n * n);
    }
//    ~Matrix() {
//        free(arr);
//        cudaFree(arr_dev);
//    }
    void del() {
        free(arr);
        hipFree(arr_dev);
    }

    double get(int row, int column) {
        return arr[row * n + column];
    }
    __device__ double get_dev(int row, int column) {
        return arr_dev[row * n + column];
    }

    void set(int row, int column, double data) {
        arr[row * n + column] = data;
    }

    void toGPU() {
        hipMemcpy(arr_dev, arr, sizeof(double)*9, hipMemcpyHostToDevice);
    }

    void toCPU() {
        hipMemcpy(arr, arr_dev, sizeof(double) * 9, hipMemcpyDeviceToHost);
    }

    void inverse() {
        double det = (arr[0] * arr[4] * arr[8]) +
                     (arr[1] * arr[5] * arr[6]) +
                     (arr[2] * arr[3] * arr[7]) -
                     (arr[2] * arr[4] * arr[6]) -
                     (arr[1] * arr[3] * arr[8]) -
                     (arr[0] * arr[5] * arr[7]);

        double * temp_matrix = (double * ) malloc(sizeof(double) * 9);

        temp_matrix[0] =   arr[4] * arr[8] - arr[5] * arr[7];
        temp_matrix[1] = -(arr[1] * arr[8] - arr[2] * arr[7]);
        temp_matrix[2] =   arr[1] * arr[5] - arr[2] * arr[4];
        temp_matrix[3] = -(arr[3] * arr[8] - arr[5] * arr[6]);
        temp_matrix[4] =   arr[0] * arr[8] - arr[2] * arr[6];
        temp_matrix[5] = -(arr[0] * arr[5] - arr[2] * arr[3]);
        temp_matrix[6] =   arr[3] * arr[7] - arr[4] * arr[6];
        temp_matrix[7] = -(arr[0] * arr[7] - arr[1] * arr[6]);
        temp_matrix[8] =   arr[0] * arr[4] - arr[1] * arr[3];

        for (int i = 0; i < 9; i++) {
            temp_matrix[i] /= det;
        }

        memcpy(arr, temp_matrix, 9 * sizeof(double));
        free(temp_matrix);
    }
};

struct pixel {
    unsigned char x;
    unsigned char y;
    unsigned char z;
    unsigned char a;
} typedef pixel;

// Перемножение двух матриц
void mult(double** a, int r1, int c1, double** b, int r2, int c2, double** res) {
    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c2; j++) {
            res[i][j] = 0;
            for (int k = 0; k < c1; k++)
                res[i][j] += a[i][k] * b[k][j];
        }
    }
}

// Перемножение двух векторов
void multVectors(double* a, double*b, double** c) {
    for (int i = 0; i < 3; i ++) {
        for (int j = 0 ; j < 3; j++) {
            c[i][j] = a[i] * b[j];
        }
    }
}

// Вывод матрицы в консоль
void printMatrix(double** a) {
    for (int i = 0; i < 3; i++){
        for (int j = 0; j < 3; j++){
            printf("%lf\t", a[i][j]);
        }
        printf("\n");
    }
}

void pixelToMatrix(pixel p, double ** res) {
//    double** res = init();
    res[0][0] = p.x;
    res[1][0] = p.y;
    res[2][0] = p.z;
}

// Транспонируем
void transpose(double** a, double** res) {
    for(int i=0;i<3;i++)
        for(int j=0;j<3;j++)
            res[j][i]=a[i][j];
}

// инициализирует пустую матрицу
double** init(){
    double** arr = (double**)malloc(sizeof(double*)*3);
    for(int i = 0; i < 3; i++){
        arr[i] = (double*)malloc(sizeof(double)*3);
    }
    return arr;
}

// Записывает резульат в а
void difference(double** a, double** b) {
    for(int i=0;i<3;i++)// Транспонируем
        for(int j=0;j<3;j++)
            a[i][j] -= b[i][j];
}

// Записывает резульат в а
void sum(double** a, double** b) {
    for(int i=0; i<3; i++)// Транспонируем
        for(int j=0; j<3; j++)
            a[i][j] += b[i][j];
}

__global__
void mahalanobis_distance(pixel* image_data, Matrix* avg, Matrix* cov, int pix_len, int num_classes) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x * blockDim.x;

    for (;idx < pix_len; idx += offset) {
        pixel pix = image_data[idx];
        int maxInd = 0;
        auto maxVal = (-1) * DBL_MAX;
        for (int i = 0; i < num_classes; i++) {
            double a = pix.x - avg[i].get_dev(0,0);
            double b = pix.y - avg[i].get_dev(1,0);
            double c = pix.z - avg[i].get_dev(2,0);

            double res = (-a*cov[i].get_dev(0,0) - b*cov[i].get_dev(1,0)-c*cov[i].get_dev(2,0)) * a +
                         (-a*cov[i].get_dev(0,1) - b*cov[i].get_dev(1,1)-c*cov[i].get_dev(2,1)) * b +
                         (-a*cov[i].get_dev(0,2) - b*cov[i].get_dev(1,2)-c*cov[i].get_dev(2,2)) * c;

            if (res > maxVal) {
                maxInd = i;
                maxVal = res;
            }
        }
        int a = maxInd;
        image_data[idx].a = a;
    }
}
int main() {

    // Считывание данных
    char fileIn[1024];
    char fileOut[1024];
    scanf("%s", fileIn);
    scanf("%s", fileOut);

    int** data;
    int n;
    scanf("%d", &n);
    data = (int**)malloc(sizeof(int*)*n);
    int* lens = (int*)malloc(sizeof(int)*n);

    for (int i = 0; i < n; i++) {
        scanf("%d", lens + i);
        data[i] = (int*)malloc(sizeof(int)*lens[i]*2);
        for (int j = 0; j < lens[i]*2; j++) {
            scanf("%d", data[i]+j);
        }
    }

    FILE* fp = fopen(fileIn, "rb");
    if (!fp) perror("fopen");
    int width;
    int height;

    fread(&width, sizeof(int), 1, fp);
    fread(&height, sizeof(int), 1, fp);
    auto image = (pixel*)malloc(sizeof(pixel) * width * height);
    fread(image, sizeof(pixel), width * height, fp);
    fclose(fp);

    auto avgList = (Matrix*)malloc(sizeof(Matrix) * n);
    auto covList = (Matrix*)malloc(sizeof(Matrix) * n);

    for (int i = 0; i < n; i++) {
        //AVG
        double xSum = 0;
        double ySum = 0;
        double zSum = 0;
        for (int j = 0; j < lens[i] * 2; j += 2) {
            pixel pix = image[data[i][j + 1] * width + data[i][j]];
            xSum += pix.x;
            ySum += pix.y;
            zSum += pix.z;
        }
        avgList[i] = Matrix();
        avgList[i].set(0,0, xSum * 1.0/lens[i]);
        avgList[i].set(1,0, ySum * 1.0/lens[i]);
        avgList[i].set(2,0, zSum * 1.0/lens[i]);

        avgList[i].toGPU();

        // Ковариация
        auto cov = init();
        auto left = init();
        auto right = init();
        double** transposed = init();
        covList[i] = Matrix();
        for (int j = 0; j < lens[i] * 2; j += 2) {
            pixel pix = image[data[i][j + 1] * width + data[i][j]];
            pixelToMatrix(pix, left);
            pixelToMatrix(pix, right);
            left[0][0] -= avgList[i].get(0,0);
            left[1][0] -= avgList[i].get(1,0);
            left[2][0] -= avgList[i].get(2,0);

            right[0][0] -= avgList[i].get(0,0);
            right[1][0] -= avgList[i].get(1,0);
            right[2][0] -= avgList[i].get(2,0);

            transpose(right, transposed);
            mult(left,3,1,transposed,1,3, cov);

            for (int r = 0; r < 3; r++) {
                for (int c = 0; c < 3; c++) {
                    covList[i].set(r,c, covList[i].get(r,c) + cov[r][c]);
                }
            }
        }

        free(transposed);
        free(left);
        free(right);
        free(cov);

        for(int c=0; c<3; c++)
            for(int v=0; v<3; v++)
                covList[i].set(c,v, covList[i].get(c,v) / (lens[i]-1));

        covList[i].inverse();
        covList[i].toGPU();
    }
    Matrix* covList_dev;
    hipMalloc(&covList_dev, sizeof(Matrix)  * n);
    hipMemcpy(covList_dev, covList, sizeof(Matrix) * n, hipMemcpyHostToDevice);

    Matrix* avgList_dev;
    hipMalloc(&avgList_dev, sizeof(Matrix)  * n);
    hipMemcpy(avgList_dev, avgList, sizeof(Matrix) * n, hipMemcpyHostToDevice);

    pixel * img_dev;
    hipMalloc(&img_dev, sizeof(pixel)  * width*height);
    hipMemcpy(img_dev, image, sizeof(pixel)  * width*height, hipMemcpyHostToDevice);

    mahalanobis_distance <<< 32, 32 >>> (img_dev, avgList_dev, covList_dev, width*height, n);

    hipMemcpy(image, img_dev, sizeof(pixel)  * width*height, hipMemcpyDeviceToHost);

    fp = fopen(fileOut, "wb");
    fwrite(&width , sizeof(int), 1, fp);
    fwrite(&height, sizeof(int), 1, fp);
    fwrite(image,sizeof(pixel), width * height, fp);
    fclose(fp);

    hipFree(covList_dev);
    hipFree(avgList_dev);
    hipFree(img_dev);
    for (int i = 0; i < n; i++) {
        free(data[i]);
        covList[i].del();
        avgList[i].del();
    }
    free(image);
    free(avgList);
    free(covList);
    free(lens);

    free(data);


    return 0;
}