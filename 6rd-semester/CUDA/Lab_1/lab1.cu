#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>

__global__ void pow(double *arr, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = gridDim.x * blockDim.x;

  while (idx < n) {
    arr[idx] *= arr[idx];
    idx += offset;
  }
}

int main() {
  double *arr;
  double *arr_dev;
  int n = 0;

  scanf("%d", &n);
  double* arr = (double*)malloc(n*sizeof(double));
  for (int i = 0; i < n; i++) {
    scanf("%lf", &arr[i]);
    //arr[i] = i;
  }


  hipMalloc(&arr_dev, sizeof(double)*n);
  hipMemcpy(arr_dev, arr, sizeof(double)*n, hipMemcpyHostToDevice);

  pow<<<1024, 1024>>>(arr_dev, n);

  hipMemcpy(arr, arr_dev, sizeof(double) * n, hipMemcpyDeviceToHost);

  for (int i = 0; i < n; i++) {
    printf("%e ", arr[i]);
  }
  printf("\n");

  hipFree(arr_dev);
  free(arr);

  return 0;
}