#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CSC(call)  									\
do {											\
	hipError_t res = call;							\
	if (res != hipSuccess) {							\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);								\
	}										\
} while(0)


texture<uchar4, 2, hipReadModeElementType> tex;
const double red = 0.299;
const double green = 0.587;
const double blue = 0.114;


__global__ void roberts(uchar4 *out, int w, int h) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  int offsetx = blockDim.x * gridDim.x;
  int offsety = blockDim.y * gridDim.y;
  int x, y;
  uchar4 p,p1,p2,p3;
  for(y = idy; y < h; y += offsety)
    for(x = idx; x < w; x += offsetx) {
      p  = tex2D(tex, x+1, y+1);
      p1 = tex2D(tex, x,   y);
      p2 = tex2D(tex, x+1, y);
      p3 = tex2D(tex, x,   y+1);

      double Y  = red * p.x  + green * p.y  + blue * p.z;
      double Y1 = red * p1.x + green * p1.y + blue * p1.z;
      double Y2 = red * p2.x + green * p2.y + blue * p2.z;
      double Y3 = red * p3.x + green * p3.y + blue * p3.z;
      double Yres = sqrt((Y-Y1)*(Y-Y1) + (Y2-Y3)*(Y2-Y3));
      if (Yres > 255)
        Yres = 255;

      out[y * w + x] = make_uchar4(Yres, Yres, Yres, p.w);
    }
}
// Y = 0.299 R+ 0.587*G + 0.144*B

int main() {
  char* inFileName = (char*)malloc(1024);
  char* outFileName = (char*)malloc(1024);
  scanf("%s", inFileName);
  scanf("%s", outFileName);

  int w, h;
  FILE *fp = fopen(inFileName, "rb");
  fread(&w, sizeof(int), 1, fp);
  fread(&h, sizeof(int), 1, fp);
  uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
  fread(data, sizeof(uchar4), w * h, fp);
  fclose(fp);

  hipArray *arr;
  hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
  CSC(hipMallocArray(&arr, &ch, w, h));

  CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

  tex.normalized = false;
  tex.filterMode = hipFilterModePoint;
  tex.channelDesc = ch;
  tex.addressMode[0] = hipAddressModeClamp;
  tex.addressMode[1] = hipAddressModeClamp;

  CSC(hipBindTextureToArray(tex, arr, ch));

  uchar4 *dev_out;
  CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

  roberts<<< dim3(16, 16), dim3(32, 32) >>>(dev_out, w, h);
  CSC(hipGetLastError());

  CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
  CSC(hipUnbindTexture(tex));
  CSC(hipFreeArray(arr));
  CSC(hipFree(dev_out));

  fp = fopen(outFileName, "wb");
  fwrite(&w, sizeof(int), 1, fp);
  fwrite(&h, sizeof(int), 1, fp);
  fwrite(data, sizeof(uchar4), w * h, fp);
  fclose(fp);

  free(data);
  free(inFileName);
  free(outFileName);
  return 0;
}
